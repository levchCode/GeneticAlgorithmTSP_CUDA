#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <time.h>

#include <stdio.h>

#include "genetic_algorithm.h"

template <class T> void swap (T& a, T& b){
	T c(a); a = b; b = c;
}

__device__ int rand_int(int *seed){
	unsigned int xi = *(unsigned int *)seed;
	unsigned int m = 65537 * 67777;

	xi = (xi * xi) % m;
	*seed = *(unsigned int *)&xi;

	return xi % RAND_MAX_GA;
}

__device__ float rand_float(int *seed){
	float r = (float)(rand_int(seed) % 100);
	return r / 100.0;
}

__device__ int selectSpecimen(specimen *pop, int size, int *random_seed){
	int i, j;
	i = rand_int(random_seed) % size;
	j = (rand_int(random_seed) % (size - 1) + i + 1) % size;

	return (pop[i].fitness < pop[j].fitness) ? i : j;
}

__device__ void crossover(specimen *parent, specimen *offspring, int *random_seed){
	int i;
	int cpoint = rand_int(random_seed) % specimenbits;
	for(i = 0; i < specimenbits; ++i){
		int part = (i < cpoint) ? 1 : 0;
		offspring[0].p[i] = parent[part].p[i];
		offspring[1].p[i] = parent[1-part].p[i];
		offspring[0].q[i] = parent[part].q[i];
		offspring[1].q[i] = parent[1 - part].q[i];
	}

	offspring[0].fitness = 0;
	offspring[1].fitness = 0;
}

__device__ void mutate(specimen *parent, int *random_seed){
	int i;

	int mp = rand_int(random_seed) % 100 - 50;
	int mq = rand_int(random_seed) % 100 - 50;

	for(i = 0; i < specimenbits; ++i){
		if(rand_float(random_seed) < pmutation){
			parent->p[i] = mp + parent->p[i];
			parent->q[i] = mq + parent->q[i];
		}
	}
}

__device__ __host__ int sum(const unsigned int *m)
	{
		int s = 0;
		int i = 0;
		for (i = 0; i < specimenbits; ++i) {
			s += (int)(m[i]);
		}
		return s;
	}

__device__ float fitness(const specimen *sp){

	return abs(N - sum(sp->p)*sum(sp->q));
}

__global__ void initPopulation(specimen *pop, const int size, const int random_seed){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < size){
		int seed = random_seed + i, j;
		for(j = 0; j < specimenbits; ++j)
			pop[i].p[j] = rand_int(&seed) % 10;
			pop[i].q[j] = rand_int(&seed) % 10;
	}
}

__global__ void newGeneration(specimen *pop, specimen *newpop, const int size, const int random_seed){
	const int i = 2 * (blockIdx.x*blockDim.x + threadIdx.x);
	if((i + 1) >= size) return;

	specimen parent[2], offspring[2];
	int seed = random_seed + i;

	parent[0] = pop[selectSpecimen(pop, size, &seed)];
	parent[1] = pop[selectSpecimen(pop, size, &seed)];

	if(rand_float(&seed) < pcross){
		crossover(parent, offspring, &seed);
	} else {
		offspring[0] = parent[0];
		offspring[1] = parent[1];
	}

	mutate(&offspring[0], &seed);
	mutate(&offspring[1], &seed);
	newpop[i] = offspring[0];
	newpop[i+1] = offspring[1];
}

__global__ void countFitness(specimen *pop, const int size){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < size){
		const specimen sp = pop[i];
		pop[i].fitness = fitness(&sp);
	}
}

__global__ void findBestSpecimen(specimen *pop, const int size){
	const int index = threadIdx.x;
	if(index >= THREADS) return;

	int bestIndex = index, i;
	for(i = index+THREADS; i < size; i += THREADS){
		if(pop[bestIndex].fitness < pop[i].fitness)
			bestIndex = i;
	}

	__shared__ int buffer[THREADS];
	buffer[index] = bestIndex;
	__syncthreads();

	if(index == 0){
		for(i = 0; i < THREADS; ++i)
			if(pop[bestIndex].fitness < pop[ buffer[i] ].fitness)
				bestIndex = buffer[i];

		pop[0] = pop[bestIndex];
	}
}



int main(){
	srand (time(NULL));

	const int population = THREADS * BLOCKS;
	specimen best;

	specimen *devPopulation = 0, *devNewPopulation = 0;
	hipMalloc((void**)&devPopulation, sizeof(specimen) * population);
	hipMalloc((void**)&devNewPopulation, sizeof(specimen) * population);

	initPopulation<<<BLOCKS, THREADS>>>(devPopulation, population, rand() % RAND_MAX_GA);
	hipDeviceSynchronize();

	while(true){

		findBestSpecimen << <1, THREADS >> >(devPopulation, population);
		hipDeviceSynchronize();

		hipMemcpy(&best, &devPopulation[0], sizeof(specimen), hipMemcpyDeviceToHost);

		int p = sum(best.p);
		int q = sum(best.q);

		if (p*q == N) {
			printf("Found: p = %d q = %d", p, q);
			break;
		}

		printf("Best fitness: %f (p = %d | q = %d)", best.fitness, p, q);
		printf("\n");

		countFitness<<<BLOCKS, THREADS>>>(devPopulation, population);
		newGeneration<<<BLOCKS, HALF_THREADS>>>(devPopulation, devNewPopulation, population, rand() % RAND_MAX_GA);
		hipDeviceSynchronize();
		swap(devPopulation, devNewPopulation); 
	}


	

	getchar();

	hipFree(devPopulation);
	hipFree(devNewPopulation);

	
}